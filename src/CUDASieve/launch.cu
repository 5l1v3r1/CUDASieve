#include "hip/hip_runtime.h"
/*

launch.cu

Host functions for CUDASieve which interface with the device
Curtis Seizert  <cseizert@gmail.com>

The naming convention for sieve sizes:
 sieveWords == number of 32-bit integers in the array
 sieveBits == number of total bits in the array (i.e. words * 32)
 sieveSpan == numbers covered by the sieve, since only odds are being sieved
              this means bits * 2

*/
#include "CUDASieve/host.hpp"
#include "CUDASieve/cudasieve.hpp"
#include "CUDASieve/global.cuh"
#include "CUDASieve/launch.cuh"

#include <iostream>
#include <ctime>
#include <cmath>

/*
                      *************************
                      ****** PrimeOutList *****
                      *************************

PrimeOutList is the class that deals with getting lists of primes from pre-existing
sieve arrays.

*/

PrimeOutList::PrimeOutList(CudaSieve & sieve)
{
  blocks = (sieve.bigsieve.bigSieveBits)/(32*PL_SIEVE_WORDS);
  threads = 512;

  hist_size_lg = blocks/512 + 1;

  allocateDevice();
}

inline void PrimeOutList::allocateDevice()
{
  d_histogram =       safeCudaMalloc(d_histogram, blocks*sizeof(uint32_t));
  d_histogram_lg =    safeCudaMalloc(d_histogram_lg, hist_size_lg*sizeof(uint32_t));

  hipMemset(d_histogram, 0, blocks*sizeof(uint32_t));
  hipMemset(d_histogram_lg, 0, hist_size_lg*sizeof(uint32_t));
}

inline void PrimeOutList::fetch(BigSieve & bigsieve, CudaSieve & sieve)
{
  uint64_t * d_ptr = sieve.d_primeOut + * sieve.kerneldata.h_count;

  hipMemset(d_histogram, 0, blocks*sizeof(uint32_t));
  hipMemset(d_histogram_lg, 0, hist_size_lg*sizeof(uint32_t));

  device::makeHistogram_PLout<<<bigsieve.bigSieveKB, THREADS_PER_BLOCK>>>
    (bigsieve.d_bigSieve, d_histogram, bigsieve.bottom, bigsieve.top);
  device::exclusiveScan<<<hist_size_lg,threads,threads*sizeof(uint32_t)>>>
    (d_histogram, d_histogram_lg, blocks);
  device::exclusiveScan<<<1,hist_size_lg,hist_size_lg*sizeof(uint32_t)>>>
    (d_histogram_lg, sieve.kerneldata.d_count, hist_size_lg);
  device::increment<<<hist_size_lg,threads,threads*sizeof(uint32_t)>>>
    (d_histogram, d_histogram_lg, blocks);
  device::makePrimeList_PLout<<<bigsieve.bigSieveKB, THREADS_PER_BLOCK>>>
    (d_ptr, d_histogram, bigsieve.d_bigSieve, bigsieve.bottom, bigsieve.top);
}

PrimeOutList::~PrimeOutList()
{
  safeCudaFree(d_histogram);
  safeCudaFree(d_histogram_lg);
}

/*
                        **************************
                        ******* PrimeList ********
                        **************************

PrimeList is the class that makes a list of sieving primes on the device.  This work is orchestrated
by the static function PrimeList::getSievingPrimes(...), which returns a device pointer.
*/

uint32_t * PrimeList::getSievingPrimes(uint32_t maxPrime, uint32_t & primeListLength, bool silent)
{
  PrimeList primelist(maxPrime);

  primelist.allocate();
  primelist.iterSieve();
  primeListLength = (uint32_t)* primelist.kerneldata.h_count;
  if(!silent) std::cout << "List of sieving primes in " << primelist.timer.get_ms() << " ms." << std::endl;
  uint32_t * temp = primelist.d_primeList;
  primelist.d_primeList = NULL;

  return temp;
}

PrimeList::PrimeList(uint32_t maxPrime)
{
  this -> maxPrime = maxPrime;
  if(maxPrime < pow(2,22)) bigSieveKB = 256;
  if(maxPrime > pow(2,30)) bigSieveKB = 16384;
  if(maxPrime > pow(2,31)) bigSieveKB = 32768;

  blocks = (bigSieveKB << 13)/(32*PL_SIEVE_WORDS);
  threads = 512;

  hist_size_lg = blocks/512 + 1;
  piHighGuess = (int) (maxPrime/log(maxPrime))*(1+1.2762/log(maxPrime)); // this is an empirically derived formula to calculate a high bound for the prime counting function pi(x)

  PL_Max = std::min((uint32_t)65536, maxPrime);
}

void PrimeList::allocate()
{
  kerneldata.allocate();

  d_bigSieve =        safeCudaMalloc(d_bigSieve, bigSieveKB*256*sizeof(uint32_t));
  d_primeList =       safeCudaMalloc(d_primeList, piHighGuess*sizeof(uint32_t));
  d_histogram =       safeCudaMalloc(d_histogram, blocks*sizeof(uint32_t));
  d_histogram_lg =    safeCudaMalloc(d_histogram_lg, hist_size_lg*sizeof(uint32_t));

  hipMemset(d_histogram, 0, blocks*sizeof(uint32_t));
  hipMemset(d_histogram_lg, 0, hist_size_lg*sizeof(uint32_t));

  hipMemset(d_primeList, 0, piHighGuess*sizeof(uint32_t));

}

void PrimeList::iterSieve() // makes the list of primes on the device and then copies them back to the host
{
  timer.start();

  hipMemset(d_bigSieve, 0, bigSieveKB*256*sizeof(uint32_t));

  device::firstPrimeList<<<1, 256>>>(d_primeList, kerneldata.d_count, 32768, PL_Max);
  hipDeviceSynchronize();
  primeListLength = (uint32_t)* kerneldata.h_count;
  if(maxPrime > PL_Max){

    for(uint64_t bottom = 65536; bottom < maxPrime; bottom += (bigSieveKB << 14)){

      hipMemset(d_histogram, 0, blocks*sizeof(uint32_t));
      hipMemset(d_histogram_lg, 0, hist_size_lg*sizeof(uint32_t));

      device::bigSieveSm<<<bigSieveKB/sieveKB, THREADS_PER_BLOCK, (sieveKB << 10)>>>
        (d_primeList, d_bigSieve, bottom, sieveKB, primeListLength);

      uint32_t * d_ptr = d_primeList + * kerneldata.h_count;

      hipDeviceSynchronize();

      device::makeHistogram_PLout<<<bigSieveKB, THREADS_PER_BLOCK>>>
        (d_bigSieve, d_histogram, (uint64_t)bottom, (uint64_t) maxPrime);
      device::exclusiveScan<<<hist_size_lg,threads,threads*sizeof(uint32_t)>>>
        (d_histogram, d_histogram_lg, blocks);
      device::exclusiveScan<<<1,hist_size_lg,hist_size_lg*sizeof(uint32_t)>>>
        (d_histogram_lg, kerneldata.d_count, hist_size_lg);
      device::increment<<<hist_size_lg,threads,threads*sizeof(uint32_t)>>>
        (d_histogram, d_histogram_lg, blocks);
      device::makePrimeList_PLout<<<bigSieveKB, THREADS_PER_BLOCK>>>
        (d_ptr, d_histogram, d_bigSieve, bottom, maxPrime);
      hipDeviceSynchronize();
    }
  }
  timer.stop();
}

PrimeList::~PrimeList()
{
  hipFree(d_bigSieve);
  hipFree(d_histogram);
  hipFree(d_histogram_lg);
}

void SmallSieve::run(CudaSieve & sieve)
{
  if(!sieve.flags[0])                   sieve.smallsieve.count(sieve);
  if(!sieve.flags[30])                  sieve.smallsieve.timer.displayTime();
}

void SmallSieve::createStreams() // this takes about 0.025 ms
{
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);
  hipStreamCreate(&stream[2]);
}

void SmallSieve::count(CudaSieve & sieve)
{
  createStreams();
  timer.start();
  device::smallSieve<<<totBlocks, THREADS_PER_BLOCK, (sieve.sieveKB << 10), stream[0]>>>
    (sieve.d_primeList, sieve.kerneldata.d_count, kernelBottom, sieve.sieveBits, sieve.primeListLength, sieve.kerneldata.d_blocksComplete);
  if(sieve.isFlag(4)) device::smallSieveIncompleteTop<<<1, THREADS_PER_BLOCK, 0, stream[1]>>>
    (sieve.d_primeList, top, sieve.sieveBits, sieve.primeListLength, sieve.top, sieve.kerneldata.d_count, sieve.kerneldata.d_blocksComplete, 1);
  if(sieve.isFlag(5)) device::smallSieveIncompleteTop<<<1, THREADS_PER_BLOCK, 0, stream[2]>>>
    (sieve.d_primeList, kernelBottom, sieve.sieveBits, sieve.primeListLength, sieve.bottom-1, sieve.kerneldata.d_count, sieve.kerneldata.d_blocksComplete, 0);
  if(!sieve.isFlag(30)) sieve.kerneldata.displayProgress(totBlocks+sieve.isFlag(4)+sieve.isFlag(5));
  hipDeviceSynchronize();
  timer.stop();
}

void BigSieve::run(CudaSieve & sieve) // coordinates the functions of this class for the CLI
{
  sieve.bigsieve.setParameters(sieve);
  sieve.bigsieve.allocate();

  sieve.bigsieve.fillNextMult();

  if(!sieve.flags[30])                      host::displayAttributes(sieve.bigsieve);

  if(sieve.flags[0]   && !sieve.flags[2])   sieve.bigsieve.launchLoopPrimesSmall(sieve);
  if(sieve.flags[0]   &&  sieve.flags[2])   sieve.bigsieve.launchLoopPrimes(sieve);
  if(!sieve.flags[0])                       sieve.bigsieve.launchLoop(sieve);
  if(!sieve.flags[30])                      sieve.bigsieve.timer.displayTime();
}

BigSieve::BigSieve(CudaSieve & sieve)
{
  setParameters(sieve);
  allocate();
}

void BigSieve::setParameters(CudaSieve & sieve)
{
  // Copy relevant sieve paramters
  sieveKB = 32;                                        // this is the optimal value for the big sieve
  if(!sieve.flags[1]) this -> sieveKB = sieve.sieveKB; // this defaults to 16, which is faster < 2**40
  this -> primeListLength = sieve.primeListLength;
  this -> d_primeList = sieve.d_primeList;
  this -> top = sieve.top;
  silent = sieve.flags[30];

  // Calculate BigSieve specific parameters
  bigSieveBits = bigSieveKB << 13;
  blocksSm = bigSieveKB/sieveKB;
  blocksLg = primeListLength/THREADS_PER_BLOCK_LG;
  log2bigSieveSpan = log2((double) bigSieveBits) + 1;
  if(!sieve.flags[0])   this -> bottom = max((1ull << 40), (unsigned long long) sieve.bottom);
  else                  this -> bottom = sieve.bottom;
  cutoff = bottom;
  bottom -= bottom%64;
  totIter = (this->top-this->bottom)/(2*this->bigSieveBits);
}

void BigSieve::allocate()
{
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);

  d_next =      safeCudaMalloc(d_next, primeListLength*sizeof(uint32_t));
  d_away =      safeCudaMalloc(d_away, primeListLength*sizeof(uint16_t));
  d_bigSieve =  safeCudaMalloc(d_bigSieve, bigSieveKB*256*sizeof(uint32_t));

  hipMemset(d_bigSieve, 0, bigSieveKB*256*sizeof(uint32_t));
}

void BigSieve::fillNextMult()
{
  timer.start();

  device::getNextMult30<<<blocksLg+1,THREADS_PER_BLOCK_LG>>>
    (d_primeList, d_next, d_away, primeListLength, bottom, bigSieveBits, log2bigSieveSpan);

  timer.stop();
  time_ms = timer.get_ms();
  hipDeviceSynchronize();
}

/*
for BigSieve, kernels are launched iteratively.  "bigSieveSm" is essentially the same as
the small sieve SMem kernel (32 kb sieving array in L1), except that it only sieves with the first 65536
primes and copies its output (in an atomicOr operation) to a sieve in global memory that is launched
concurrently.  That sieve is "bigSieveLg," and it sieves with the remaining primes using
Oliveira's bucket method (described in global.cu) with a much larger sieve array (1024 -
4096 kb stored in global memory).  At the end of the operation of these two kernels, the large
global mem sieve has all the composites crossed off, and is counted and zeroed with bigSieveCount.
*/

void BigSieve::launchLoop(CudaSieve & sieve) // for CLI
{
  timer.start();
  if(totIter > 0){
    for(uint64_t value = 1; bottom + 2* bigSieveBits <= top; bottom += 2*bigSieveBits, value++){
      hipDeviceSynchronize();

      device::bigSieveSm<<<blocksSm, THREADS_PER_BLOCK, (sieveKB << 10), stream[0]>>>
        (d_primeList, d_bigSieve, bottom, sieveKB, 65536);
      device::bigSieveLg<<<blocksLg, THREADS_PER_BLOCK_LG, 0, stream[1]>>>
        (d_primeList, d_next, d_away, d_bigSieve, bigSieveBits, primeListLength, log2bigSieveSpan);

      if(bottom < cutoff){
        hipDeviceSynchronize();
        device::zeroBottomWord<<<1,1,0,stream[1]>>>(d_bigSieve, bottom, cutoff);
        }
      hipDeviceSynchronize();
      device::bigSieveCount<<<blocksSm, THREADS_PER_BLOCK, (THREADS_PER_BLOCK*sizeof(uint32_t)), stream[0]>>>
        (d_bigSieve, sieveKB, sieve.kerneldata.d_count);

      if(!silent) sieve.kerneldata.displayProgress(value, totIter);
    }
  }
  if(bottom < top) countPartialTop(sieve);
  timer.stop();
  if(!silent) sieve.kerneldata.displayProgress(1, 1);
}

void BigSieve::countPartialTop(CudaSieve & sieve)
{
  hipDeviceSynchronize();

  device::bigSieveSm<<<blocksSm, THREADS_PER_BLOCK, (sieveKB << 10), stream[0]>>>
    (d_primeList, d_bigSieve, bottom, sieveKB, 65536);
  device::bigSieveLg<<<blocksLg, THREADS_PER_BLOCK_LG, 0, stream[1]>>>
    (d_primeList, d_next, d_away, d_bigSieve, bigSieveBits, primeListLength, log2bigSieveSpan);

  hipDeviceSynchronize();

  if(bottom < cutoff){
    hipDeviceSynchronize();
    device::zeroBottomWord<<<1,1,0,stream[1]>>>(d_bigSieve, bottom, cutoff);
    }
  device::bigSieveCountPartial<<<blocksSm, THREADS_PER_BLOCK, (THREADS_PER_BLOCK*sizeof(uint32_t))>>>
    (d_bigSieve, sieveKB, bottom, top, sieve.kerneldata.d_count);

  hipDeviceSynchronize();
}

/*
this is only used for debugging at present.  It copies the bitsieve back to the host after
each iteration and increments the host pointer to the end of the data copied back.  This
gives a 'compressed' set of all the primes generated by the sieve.  An equivalent data set
can be generated from the output of a different prime number generator, and the sets can
be compared through various bitwise operations.  This is how the prime output of CUDASieve
is checked against primesieve.
*/
void BigSieve::launchLoopCopy(CudaSieve & sieve)
{
  timer.start();
  sieve.allocateSieveOut((top-bottom)/16);
  this -> ptr32 = sieve.sieveOut;
  for(uint64_t value = 1; bottom + 2* bigSieveBits <= top; bottom += 2*bigSieveBits, value++){

    device::bigSieveSm<<<blocksSm, THREADS_PER_BLOCK, (sieveKB << 10), stream[0]>>>
      (d_primeList, d_bigSieve, bottom, sieveKB, 65536);
    if(primeListLength > 65536) device::bigSieveLg<<<blocksLg, THREADS_PER_BLOCK_LG, 0, stream[1]>>>
      (d_primeList, d_next, d_away, d_bigSieve, bigSieveBits, primeListLength, log2bigSieveSpan);

    hipDeviceSynchronize();

    hipMemcpy(ptr32, d_bigSieve, bigSieveKB*1024, hipMemcpyDeviceToHost); // copy global mem sieve to appropriate
                                                                            // elements of host bitsieve output
    ptr32 +=  bigSieveKB*256;                                               // increment pointer

    device::bigSieveCount<<<blocksSm, THREADS_PER_BLOCK, (THREADS_PER_BLOCK*sizeof(uint32_t))>>>
      (d_bigSieve, sieveKB, sieve.kerneldata.d_count);                            // count and zero
  }
  timer.stop();
  if(!silent) sieve.kerneldata.displayProgress(totIter, totIter);
}

void BigSieve::launchLoopPrimes(CudaSieve & sieve) // makes the list of primes on the device and then copies them back to the host
{
  PrimeOutList newlist(sieve);

  timer.start();

  for(uint64_t value = 1; bottom < top; bottom += 2*bigSieveBits, value++){

    device::bigSieveSm<<<blocksSm, THREADS_PER_BLOCK, (sieveKB << 10), stream[0]>>>
      (d_primeList, d_bigSieve, bottom, sieveKB, 65536);
    device::bigSieveLg<<<blocksLg, THREADS_PER_BLOCK_LG, 0, stream[1]>>>
      (d_primeList, d_next, d_away, d_bigSieve,  bigSieveBits, primeListLength, log2bigSieveSpan);

    if(bottom < cutoff){
      hipDeviceSynchronize();
      device::zeroBottomWord<<<1,1,0,stream[1]>>>(d_bigSieve, bottom, cutoff);
      }

    hipDeviceSynchronize();

    newlist.fetch(*this, sieve);
    if(!silent && totIter != 0) sieve.kerneldata.displayProgress(value, max(1ul, (unsigned long)totIter));
  }
  hipDeviceSynchronize();
  timer.stop();
  if(!silent) {sieve.kerneldata.displayProgress(1, 1); std::cout<<std::endl;}
}

void BigSieve::launchLoopPrimesSmall(CudaSieve & sieve) // makes the list of primes on the device and then copies them back to the host
{
  PrimeOutList newlist(sieve);

  timer.start();

  for(uint64_t value = 1; bottom < top; bottom += 2*bigSieveBits, value++){
;
    device::bigSieveSm<<<blocksSm, THREADS_PER_BLOCK, (sieveKB << 10), stream[0]>>>
      (d_primeList, d_bigSieve, bottom, sieveKB, primeListLength);

    if(bottom < sqrt(top) && bottom != 0){ // these conditionals add <<1% time to sieves
      hipDeviceSynchronize();             // taking longer than 1 ms.
      device::zeroPrimeList<<<1,256,0,stream[1]>>>(d_bigSieve, bottom, d_primeList, primeListLength);
    }
    if(bottom < cutoff){
      hipDeviceSynchronize();
      device::zeroBottomWord<<<1,1,0,stream[1]>>>(d_bigSieve, bottom, cutoff);
      }
    hipDeviceSynchronize();

    newlist.fetch(*this, sieve);
    if(!silent) sieve.kerneldata.displayProgress(value, max(1ul, (unsigned long)totIter));
  }
  hipDeviceSynchronize();
  timer.stop();
  if(!silent) {sieve.kerneldata.displayProgress(1, 1); std::cout<<std::endl;}
}

BigSieve::~BigSieve()
{
  safeCudaFree(d_next);
  safeCudaFree(d_away);
  safeCudaFree(d_bigSieve);
}
